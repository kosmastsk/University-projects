#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <assert.h>
#include <math.h>
#include "blas.h"
#include "cuda_utils.h"



__global__ void launch_axpy_kernel(int N, float ALPHA, float *X, int INCX,  float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] += ALPHA*X[i*INCX];
}

__global__ void launch_zero_init_kernel(int N, float *X)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i] = 0.f;
}

__global__ void launch_cross_entropy_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = -truth[i]*log(pred[i]) - (1-truth[i])*log(1-pred[i]);
        delta[i] = diff;
    }
}

extern "C" void zero_init_gpu_array(int N, float * X)
{
    launch_zero_init_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X);
}

extern "C" void axpy_cuda(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    launch_axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
}


extern "C" void cross_entropy(int n, float *pred, float *truth, float *delta, float *error)
{
    launch_cross_entropy_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
}