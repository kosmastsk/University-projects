#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "activations.h"
#include "cuda_utils.h"


__device__ float sigmoid_activate_kernel(float x){return 1./(1. + exp(-x));}
__device__ float relu_activate_kernel(float x){return x*(x>0);}
__device__ float tanh_activate_kernel(float x){return (2/(1 + exp(-2*x)) - 1);}

__device__ float sigmoid_gradient_kernel(float x){return (1-x)*x;}
__device__ float relu_gradient_kernel(float x){return (x>0);}
__device__ float tanh_gradient_kernel(float x){return 1-x*x;}


__device__ float activate_kernel(float x, ACTIVATION a)
{
    switch(a){
        case SIGMOID:
            return sigmoid_activate_kernel(x);
        case RELU:
            return relu_activate_kernel(x);
        case TANH:
            return tanh_activate_kernel(x);
    }
    return 0;
}

__device__ float gradient_kernel(float x, ACTIVATION a)
{
    switch(a){
        case SIGMOID:
            return sigmoid_gradient_kernel(x);
        case RELU:
            return relu_gradient_kernel(x);
        case TANH:
            return tanh_gradient_kernel(x);
    }
    return 0;
}

// sti cuda utils 8a mpei to check error kai to cudagridsize kai to blocksize, heuristic grid_size, todo  cudaOccupancy

__global__ void activate_array_kernel(float *x, int n, ACTIVATION a)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x; // check stackoverflow
    // https://stackoverflow.com/questions/16403972/cuda-thread-addressing-threadidx-x-threadidx-y-threadidx-z-and-block-addres?rq=1
    // na ginei me for?
    if(i < n) x[i] = activate_kernel(x[i], a);
}

__global__ void gradient_array_kernel(float *x, int n, ACTIVATION a, float *delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n) delta[i] *= gradient_kernel(x[i], a);
}

extern "C" void activate_array_gpu(float *x, int n, ACTIVATION a)
{
    activate_array_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, a);
}

extern "C" void gradient_array_gpu(float *x, int n, ACTIVATION a, float *delta) {
    gradient_array_kernel << < cuda_gridsize(n), BLOCK >> > (x, n, a, delta);
}