
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Array access macros
#define INPUT(i,j) A[(i) + (j)*(m+patchSize-1)]
#define OUTPUT(i,j) B[(i) + (j)*m*m]
#define FILTER(i) H[(i)]

__global__ void sampleAdd(double const * const A, double *B, double *H, int m, int n, int patchSize) {
        // Get pixel (x,y) in input
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;

        if (i>=((patchSize - 1) / 2) && i<=m+((patchSize - 1) / 2) && j>=((patchSize - 1) / 2) && j<=m+((patchSize - 1) / 2)) { //do not scan pixels in the pad
                for (int k = -(patchSize - 1) / 2; k <= (patchSize - 1) / 2; k++) { //scan the neighbours in an area patchSize x patchSize
                        for (int l = -(patchSize - 1) / 2; l <= (patchSize - 1) / 2; l++) {
                                OUTPUT(i - ((patchSize - 1) / 2) + m * (j - ((patchSize - 1) / 2)), k + ((patchSize - 1) / 2) + (l + ((patchSize - 1) / 2)) * patchSize) = INPUT(k + i, l + j); //assign the neighbors' value
                                OUTPUT(i - ((patchSize - 1) / 2) + m * (j - ((patchSize - 1) / 2)), k + ((patchSize - 1) / 2) + (l + ((patchSize - 1) / 2)) * patchSize) *= (FILTER(k + ((patchSize - 1) / 2) + (l + ((patchSize - 1) / 2)) * patchSize)); //multiply that value with a filter
                        }
                }
        }
}
