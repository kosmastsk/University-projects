
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// Array access macros
#define b(i,j) B[(i) + (j)*m*m]

__global__ void Kernel2(double *I,double *B, int m, int n, int patchSize, double filtSigma) {
        // Get pixel (x,y) in input
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int pixelid = i+j*m;
        double sum = 0;

        if(pixelid<m*n) {
                int l=0;
                double sum2 = 0;

                for(int p = 0; p<pixelid; p++) {
                        sum2 = sum2 +  m*n-1 - p;
                }
                double sumImatrix = 0;
                double sumRows = 0;
                for(l=0; l<m*n; l++) {
                        sum = 0;
                        for(int k=0; k<patchSize*patchSize; k++) {
                                sum+=(b(pixelid,k)-b(l,k))*(b(pixelid,k)-b(l,k)); //calculate distances and add them to sum
                        }
                        sumRows +=  exp(-sqrt(sum)*sqrt(sum)/filtSigma);
                        sumImatrix += exp(-sqrt(sum)*sqrt(sum)/filtSigma)*I[l];
                }
                I[pixelid] = sumImatrix/sumRows;
        }
}
